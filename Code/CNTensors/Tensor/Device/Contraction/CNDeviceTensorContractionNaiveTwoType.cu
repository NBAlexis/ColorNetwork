#include "hip/hip_runtime.h"
//=============================================================================
// FILENAME : CNDeviceTensorContractionNaiveTwoType.cu
// 
// DESCRIPTION:
// 
//
// REVISION:
//  [26/04/2021 nbale]
//=============================================================================
#include "CNTensorsPch.h"

__BEGIN_NAMESPACE

//This is the critical specialization
__OVER_ALL_TYPE_TWO(__IMPLEMENT_ContractionNaiveTwoType)

#pragma region kernels

template <class dstT, class srcT>
__global__ void _CN_LAUNCH_BOUND
_kernel_NaiveContract(
    dstT* dst,
    const dstT* __restrict__ src1,
    const srcT* __restrict__ src2,
    UINT dstIndexStart,
    const UINT* __restrict__ dstStride,
    UINT src1IndexStart,
    UINT src2IndexStart,
    const UINT* __restrict__ srcStride,
    const UINT* __restrict__ mutipliedlengths,
    BYTE byIndexCount,
    BYTE byIndexLeft,
    UINT sumStride1,
    UINT sumStride2,
    UINT sumLength,
    UBOOL bConjugate)
{
    const UINT uiIdx = threadIdx.x + blockIdx.x * blockDim.x;
    const UINT uiIdxDst = _deviceThreadIdxToTensorIdxNaive(dstStride, dstIndexStart, mutipliedlengths, uiIdx, byIndexCount);
    UINT uiIdxSrc1 = src1IndexStart;
    UINT uiIdxSrc2 = src2IndexStart;
    _deviceThreadIdxToTensorIdxNaiveLR(uiIdxSrc1, uiIdxSrc2, byIndexLeft, srcStride, mutipliedlengths, uiIdx, byIndexCount);
    dst[uiIdxDst] = _Mul(bConjugate ? _Conj(src1[uiIdxSrc1]) : src1[uiIdxSrc1], src2[uiIdxSrc2]);
    #pragma unroll
    for (UINT i = 1; i < sumLength; ++i)
    {
        dst[uiIdxDst] = _Add(dst[uiIdxDst], _Mul(bConjugate ? _Conj(src1[uiIdxSrc1 + i * sumStride1]) : src1[uiIdxSrc1 + i * sumStride1], src2[uiIdxSrc2 + i * sumStride2]));
    }
}

template <class dstT, class srcT>
__global__ void _CN_LAUNCH_BOUND
_kernel_NaiveContract_Small(
    dstT* dst,
    const dstT* __restrict__ src1,
    const srcT* __restrict__ src2,
    UINT dstIndexStart,
    const UINT* __restrict__ dstStride,
    UINT src1IndexStart,
    UINT src2IndexStart,
    const UINT* __restrict__ srcStride,
    const UINT* __restrict__ mutipliedlengths,
    BYTE byIndexCount,
    BYTE byIndexLeft,
    UINT sumStride1,
    UINT sumStride2,
    UINT sumLength,
    UBOOL bConjugate)
{
    const UINT uiIdx = threadIdx.x + blockIdx.x * blockDim.x;
    const SWorkingIndex idx = _deviceThreadIndexToWorkIndexNavie(uiIdx, mutipliedlengths, byIndexCount);
    const UINT uiIdxDst = _deviceWorkIndexToTensorIndexNaive(idx.m_Idx, dstStride, dstIndexStart, byIndexCount);
    UINT uiIdxSrc1 = src1IndexStart;
    UINT uiIdxSrc2 = src2IndexStart;
    _deviceWorkIndexToTensorIndexNaiveLR(uiIdxSrc1, uiIdxSrc2, byIndexLeft, idx.m_Idx, srcStride, byIndexCount);
    dst[uiIdxDst] = _Mul(bConjugate ? _Conj(src1[uiIdxSrc1]) : src1[uiIdxSrc1], src2[uiIdxSrc2]);

    #pragma unroll
    for (UINT i = 1; i < sumLength; ++i)
    {
        dst[uiIdxDst] = _Add(dst[uiIdxDst], _Mul(bConjugate ? _Conj(src1[uiIdxSrc1 + i * sumStride1]) : src1[uiIdxSrc1 + i * sumStride1], src2[uiIdxSrc2 + i * sumStride2]));
    }
}

template <class dstT, class srcT>
__global__ void _CN_LAUNCH_BOUND
_kernel_NaiveContractM(
    dstT* dst,
    const dstT* __restrict__ src1,
    const srcT* __restrict__ src2,
    UINT dstIndexStart,
    const UINT* __restrict__ dstStride,
    UINT src1IndexStart,
    UINT src2IndexStart,
    const UINT* __restrict__ srcStride,
    const UINT* __restrict__ mutipliedlengths,
    BYTE byIndexCount,
    BYTE byIndexLeft,
    const UINT* __restrict__ sumStride1,
    const UINT* __restrict__ sumStride2,
    const UINT* __restrict__ sumLength,
    BYTE bySumIndexCount,
    UBOOL bConjugate)
{
    const UINT uiIdx = threadIdx.x + blockIdx.x * blockDim.x;
    const UINT uiIdxDst = _deviceThreadIdxToTensorIdxNaive(dstStride, dstIndexStart, mutipliedlengths, uiIdx, byIndexCount);
    UINT uiIdxSrc1 = src1IndexStart;
    UINT uiIdxSrc2 = src2IndexStart;
    _deviceThreadIdxToTensorIdxNaiveLR(uiIdxSrc1, uiIdxSrc2, byIndexLeft, srcStride, mutipliedlengths, uiIdx, byIndexCount);

    UINT idxCounter[_CN_CONTRACTION_INDEX_COUNT_ONE_TIME];
    UINT loop = sumLength[0];
    idxCounter[0] = 0;
    for (BYTE i = 1; i < bySumIndexCount; ++i)
    {
        loop = loop * sumLength[i];
        idxCounter[i] = 0;
    }

    UINT uiLeftIndex = 0;
    UINT uiRightIndex = 0;
    dst[uiIdxDst] = _Mul(bConjugate ? _Conj(src1[uiIdxSrc1]) : src1[uiIdxSrc1], src2[uiIdxSrc2]);
    for (UINT i = 0; i < loop; ++i)
    {
        idxCounter[0] = idxCounter[0] + 1;
        uiLeftIndex = uiLeftIndex + sumStride1[0];
        uiRightIndex = uiRightIndex + sumStride2[0];
        for (BYTE i = 0; i < bySumIndexCount; ++i)
        {
            if (idxCounter[i] > sumLength[i])
            {
                idxCounter[i] = 0;
                idxCounter[i + 1] = idxCounter[i + 1] + 1;

                uiLeftIndex = uiLeftIndex - sumStride1[i];
                uiLeftIndex = uiLeftIndex + sumStride1[i + 1];
                uiRightIndex = uiRightIndex - sumStride2[i];
                uiRightIndex = uiRightIndex + sumStride2[i + 1];
            }
            dst[uiIdxDst] = _Add(dst[uiIdxDst], _Mul(bConjugate ? _Conj(src1[uiIdxSrc1 + uiLeftIndex]) : src1[uiIdxSrc1 + uiLeftIndex], src2[uiIdxSrc2 + uiRightIndex]));
        }
    }
}

template <class dstT, class srcT>
__global__ void _CN_LAUNCH_BOUND
_kernel_NaiveContractM_Small(
    dstT* dst,
    const dstT* __restrict__ src1,
    const srcT* __restrict__ src2,
    UINT dstIndexStart,
    const UINT* __restrict__ dstStride,
    UINT src1IndexStart,
    UINT src2IndexStart,
    const UINT* __restrict__ srcStride,
    const UINT* __restrict__ mutipliedlengths,
    BYTE byIndexCount,
    BYTE byIndexLeft,
    const UINT* __restrict__ sumStride1,
    const UINT* __restrict__ sumStride2,
    const UINT* __restrict__ sumLength,
    BYTE bySumIndexCount,
    UBOOL bConjugate)
{
    const UINT uiIdx = threadIdx.x + blockIdx.x * blockDim.x;
    const SWorkingIndex idx = _deviceThreadIndexToWorkIndexNavie(uiIdx, mutipliedlengths, byIndexCount);
    const UINT uiIdxDst = _deviceWorkIndexToTensorIndexNaive(idx.m_Idx, dstStride, dstIndexStart, byIndexCount);
    UINT uiIdxSrc1 = src1IndexStart;
    UINT uiIdxSrc2 = src2IndexStart;
    _deviceWorkIndexToTensorIndexNaiveLR(uiIdxSrc1, uiIdxSrc2, byIndexLeft, idx.m_Idx, srcStride, byIndexCount);

    UINT idxCounter[_CN_CONTRACTION_INDEX_COUNT_ONE_TIME];
    UINT loop = sumLength[0];
    idxCounter[0] = 0;
    for (BYTE i = 1; i < bySumIndexCount; ++i)
    {
        loop = loop * sumLength[i];
        idxCounter[i] = 0;
    }
    
    UINT uiLeftIndex = 0;
    UINT uiRightIndex = 0;
    dst[uiIdxDst] = _Mul(bConjugate ? _Conj(src1[uiIdxSrc1]) : src1[uiIdxSrc1], src2[uiIdxSrc2]);
    for (UINT i = 0; i < loop; ++i)
    {
        idxCounter[0] = idxCounter[0] + 1;
        uiLeftIndex = uiLeftIndex + sumStride1[0];
        uiRightIndex = uiRightIndex + sumStride2[0];
        for (BYTE i = 0; i < bySumIndexCount; ++i)
        {
            if (idxCounter[i] > sumLength[i])
            {
                idxCounter[i] = 0;
                idxCounter[i + 1] = idxCounter[i + 1] + 1;

                uiLeftIndex = uiLeftIndex - sumStride1[i];
                uiLeftIndex = uiLeftIndex + sumStride1[i + 1];
                uiRightIndex = uiRightIndex - sumStride2[i];
                uiRightIndex = uiRightIndex + sumStride2[i + 1];
            }
            dst[uiIdxDst] = _Add(dst[uiIdxDst], _Mul(bConjugate ? _Conj(src1[uiIdxSrc1 + uiLeftIndex]) : src1[uiIdxSrc1 + uiLeftIndex], src2[uiIdxSrc2 + uiRightIndex]));
        }
    }
}

#pragma endregion

template<class Tdst, class Tsrc>
void CNDeviceTensorContractionNaiveTwoType<Tdst, Tsrc>::Contraction(
    Tdst* pDstBuffer,
    const Tdst* __restrict__ pSrc1Buffer,
    const Tsrc* __restrict__ pSrc2Buffer,
    UINT dstIndexStart,
    const UINT* __restrict__ dstStride,
    UINT src1IndexStart,
    UINT src2IndexStart,
    const UINT* __restrict__ src1Stride,
    const UINT* __restrict__ src2Stride,
    const UINT* __restrict__ lengths,
    BYTE byIndexCount,
    BYTE byIndexCountLeft,
    UINT uiSumLength,
    UINT uiSumIndexStride1,
    UINT uiSumIndexStride2,
    UBOOL bConjugate) const
{
    const UINT dataSize = sizeof(UINT) * byIndexCount;
    const UINT totalBufferSize = dataSize * 3;
    UINT uiBlock, uiThread;
    SimpleThreadDecompose(lengths, byIndexCount, uiBlock, uiThread);

    BYTE* deviceBuffer = appGetSmallDeviceBuffer(totalBufferSize);

    UINT* hostBuffer = (UINT*)appAlloca(dataSize);
    _memcpy_hd(deviceBuffer, dstStride, dataSize);
    for (BYTE byOrder = 0; byOrder < byIndexCount; ++byOrder)
    {
        if (byOrder < byIndexCountLeft)
        {
            hostBuffer[byOrder] = src1Stride[byOrder];
        }
        else
        {
            hostBuffer[byOrder] = src2Stride[byOrder - byIndexCountLeft];
        }
    }
    _memcpy_hd(deviceBuffer + dataSize, hostBuffer, dataSize);
    __BuildMultiplyLength(deviceBuffer + (dataSize << 1));

    __KERNALCALNAIVE(_kernel_NaiveContract,
        pDstBuffer,
        pSrc1Buffer,
        pSrc2Buffer,
        dstIndexStart,
        (UINT*)deviceBuffer,
        src1IndexStart,
        src2IndexStart,
        (UINT*)(deviceBuffer + dataSize),
        (UINT*)(deviceBuffer + (dataSize << 1)),
        byIndexCount,
        byIndexCountLeft,
        uiSumIndexStride1,
        uiSumIndexStride2,
        uiSumLength,
        bConjugate
    );
}

template<class Tdst, class Tsrc>
void CNDeviceTensorContractionNaiveTwoType<Tdst, Tsrc>::Contraction(Tdst* pDstBuffer,
    const Tdst* __restrict__ pSrc1Buffer,
    const Tsrc* __restrict__ pSrc2Buffer,
    UINT dstIndexStart,
    const UINT* __restrict__ dstStride,
    UINT src1IndexStart,
    UINT src2IndexStart,
    const UINT* __restrict__ src1Stride,
    const UINT* __restrict__ src2Stride,
    const UINT* __restrict__ lengths,
    BYTE byIndexCount,
    BYTE byIndexCountLeft,
    const UINT* __restrict__ sumLeftStride,
    const UINT* __restrict__ sumRightStride,
    const UINT* __restrict__ sumlengths,
    BYTE bySumIndexCount,
    UBOOL bConjugate) const
{
    if (bySumIndexCount > _CN_CONTRACTION_INDEX_COUNT_ONE_TIME)
    {
        appCrucial(_T("Too many contraction index at one time!! %d (max supported: %d)"), bySumIndexCount, _CN_CONTRACTION_INDEX_COUNT_ONE_TIME);
        return;
    }
    const UINT dataSize = sizeof(UINT) * byIndexCount;
    const UINT dataSizeSum = sizeof(UINT) * bySumIndexCount;
    const UINT totalBufferSize = dataSize * 3 + dataSizeSum * 3;
    UINT uiBlock, uiThread;
    SimpleThreadDecompose(lengths, byIndexCount, uiBlock, uiThread);

    BYTE* deviceBuffer = appGetSmallDeviceBuffer(totalBufferSize);

    UINT* hostBuffer = (UINT*)appAlloca(dataSize);
    _memcpy_hd(deviceBuffer, dstStride, dataSize);
    for (BYTE byOrder = 0; byOrder < byIndexCount; ++byOrder)
    {
        if (byOrder < byIndexCountLeft)
        {
            hostBuffer[byOrder] = src1Stride[byOrder];
        }
        else
        {
            hostBuffer[byOrder] = src2Stride[byOrder - byIndexCountLeft];
        }
    }
    _memcpy_hd(deviceBuffer + dataSize, hostBuffer, dataSize);
    __BuildMultiplyLength(deviceBuffer + (dataSize << 1));

    _memcpy_hd(deviceBuffer + 3 * dataSize, sumLeftStride, dataSizeSum);
    _memcpy_hd(deviceBuffer + 3 * dataSize + dataSizeSum, sumRightStride, dataSizeSum);
    _memcpy_hd(deviceBuffer + 3 * dataSize + (dataSizeSum << 1), sumlengths, dataSizeSum);

    __KERNALCALNAIVE(_kernel_NaiveContractM,
        pDstBuffer,
        pSrc1Buffer,
        pSrc2Buffer,
        dstIndexStart,
        (UINT*)deviceBuffer,
        src1IndexStart,
        src2IndexStart,
        (UINT*)(deviceBuffer + dataSize),
        (UINT*)(deviceBuffer + (dataSize << 1)),
        byIndexCount,
        byIndexCountLeft,
        (UINT*)(deviceBuffer + 3 * dataSize),
        (UINT*)(deviceBuffer + 3 * dataSize + dataSizeSum),
        (UINT*)(deviceBuffer + 3 * dataSize + (dataSizeSum << 1)),
        bySumIndexCount,
        bConjugate
    );
}



__END_NAMESPACE

//=============================================================================
// END OF FILE
//=============================================================================
