#include "hip/hip_runtime.h"
//=============================================================================
// FILENAME : CNDeviceTensorContractionNaiveTwoType.cu
// 
// DESCRIPTION:
// 
//
// REVISION:
//  [26/04/2021 nbale]
//=============================================================================
#include "CNTensorsPch.h"

__BEGIN_NAMESPACE

//This is the critical specialization
__OVER_ALL_TYPE_TWO(__IMPLEMENT_ContractionNaiveTwoType)

#pragma region kernels

template <class dstT, class srcT>
__global__ void _CN_LAUNCH_BOUND
_kernel_NaiveContract(
    dstT* dst,
    const dstT* __restrict__ src1,
    const srcT* __restrict__ src2,
    UINT dstIndexStart,
    const UINT* __restrict__ dstStride,
    UINT src1IndexStart,
    UINT src2IndexStart,
    const UINT* __restrict__ srcStride,
    const UINT* __restrict__ mutipliedlengths,
    BYTE byIndexCount,
    BYTE byIndexLeft,
    UINT sumStride1,
    UINT sumStride2,
    UINT sumLength)
{
    const UINT uiIdx = threadIdx.x + blockIdx.x * blockDim.x;
    const UINT uiIdxDst = _deviceThreadIdxToTensorIdxNaive(dstStride, dstIndexStart, mutipliedlengths, uiIdx, byIndexCount);
    UINT uiIdxSrc1 = src1IndexStart;
    UINT uiIdxSrc2 = src2IndexStart;
    _deviceThreadIdxToTensorIdxNaiveLR(uiIdxSrc1, uiIdxSrc2, byIndexLeft, srcStride, mutipliedlengths, uiIdx, byIndexCount);
    dst[uiIdxDst] = _Mul(src1[uiIdxSrc1], src2[uiIdxSrc2]);
    #pragma unroll
    for (UINT i = 1; i < sumLength; ++i)
    {
        dst[uiIdxDst] = _Add(dst[uiIdxDst], _Mul(src1[uiIdxSrc1 + i * sumStride1], src2[uiIdxSrc2 + i * sumStride2]));
    }
}

template <class dstT, class srcT>
__global__ void _CN_LAUNCH_BOUND
_kernel_NaiveContract_Small(
    dstT* dst,
    const dstT* __restrict__ src1,
    const srcT* __restrict__ src2,
    UINT dstIndexStart,
    const UINT* __restrict__ dstStride,
    UINT src1IndexStart,
    UINT src2IndexStart,
    const UINT* __restrict__ srcStride,
    const UINT* __restrict__ mutipliedlengths,
    BYTE byIndexCount,
    BYTE byIndexLeft,
    UINT sumStride1,
    UINT sumStride2,
    UINT sumLength)
{
    const UINT uiIdx = threadIdx.x + blockIdx.x * blockDim.x;
    const SWorkingIndex idx = _deviceThreadIndexToWorkIndexNavie(uiIdx, mutipliedlengths, byIndexCount);
    const UINT uiIdxDst = _deviceWorkIndexToTensorIndexNaive(idx.m_Idx, dstStride, dstIndexStart, byIndexCount);
    UINT uiIdxSrc1 = src1IndexStart;
    UINT uiIdxSrc2 = src2IndexStart;
    _deviceWorkIndexToTensorIndexNaiveLR(uiIdxSrc1, uiIdxSrc2, byIndexLeft, idx.m_Idx, srcStride, byIndexCount);
    dst[uiIdxDst] = _Mul(src1[uiIdxSrc1], src2[uiIdxSrc2]);
    #pragma unroll
    for (UINT i = 1; i < sumLength; ++i)
    {
        dst[uiIdxDst] = _Add(dst[uiIdxDst], _Mul(src1[uiIdxSrc1 + i * sumStride1], src2[uiIdxSrc2 + i * sumStride2]));
    }
}

#pragma endregion

template<class Tdst, class Tsrc>
void CNDeviceTensorContractionNaiveTwoType<Tdst, Tsrc>::Contraction(
    Tdst* pDstBuffer,
    const Tdst* __restrict__ pSrc1Buffer,
    const Tsrc* __restrict__ pSrc2Buffer,
    UINT dstIndexStart,
    const UINT* __restrict__ dstStride,
    UINT src1IndexStart,
    UINT src2IndexStart,
    const UINT* __restrict__ src1Stride,
    const UINT* __restrict__ src2Stride,
    const UINT* __restrict__ lengths,
    BYTE byIndexCount,
    BYTE byIndexCountLeft,
    UINT uiSumLength,
    UINT uiSumIndexStride1,
    UINT uiSumIndexStride2) const
{
    const UINT dataSize = sizeof(UINT) * byIndexCount;
    const UINT totalBufferSize = dataSize * 3;
    UINT uiBlock, uiThread;
    SimpleThreadDecompose(lengths, byIndexCount, uiBlock, uiThread);

    BYTE* deviceBuffer = appGetSmallDeviceBuffer(totalBufferSize);

    UINT* hostBuffer = (UINT*)appAlloca(dataSize);
    _memcpy_hd(deviceBuffer, dstStride, dataSize);
    for (BYTE byOrder = 0; byOrder < byIndexCount; ++byOrder)
    {
        if (byOrder < byIndexCountLeft)
        {
            hostBuffer[byOrder] = src1Stride[byOrder];
        }
        else
        {
            hostBuffer[byOrder] = src2Stride[byOrder - byIndexCountLeft];
        }
    }
    _memcpy_hd(deviceBuffer + dataSize, hostBuffer, dataSize);
    __BuildMultiplyLength(deviceBuffer + (dataSize << 1));

    __KERNALCALNAIVE(_kernel_NaiveContract,
        pDstBuffer,
        pSrc1Buffer,
        pSrc2Buffer,
        dstIndexStart,
        (UINT*)deviceBuffer,
        src1IndexStart,
        src2IndexStart,
        (UINT*)(deviceBuffer + dataSize),
        (UINT*)(deviceBuffer + (dataSize << 1)),
        byIndexCount,
        byIndexCountLeft,
        uiSumIndexStride1,
        uiSumIndexStride2,
        uiSumLength
    );
}


__END_NAMESPACE

//=============================================================================
// END OF FILE
//=============================================================================
